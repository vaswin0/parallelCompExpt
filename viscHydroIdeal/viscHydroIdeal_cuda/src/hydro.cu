#include "hip/hip_runtime.h"
#include "hydro.h"

#define del_eps 1e-7

hydro::hydro(EoS* _eos, grid* _f,idb* _IDB,double _tau0, double _dt, cnvrt* _CN)
{
 eos = _eos;
 f = _f;
 CN = _CN;
 tau = _tau0;
 IDB = _IDB ;
 dt =_dt;
}

hydro::~hydro()
{
}


void hydro::set_dtau(double deltaTau)
{
  dt = deltaTau;
  if (dt > IDB->dx / 2. ||
      dt > IDB->dy / 2. /*|| dt > tau*IDB->deta */) {
    cout << "too big delta_tau " << dt << "  " << IDB->dx << "  " << IDB->dy
	 << "  " << tau * IDB->deta << endl;
    exit(1);
  }
}


void hydro::evolve()
{

  for(int iy = 0; iy<f->get_ny(); iy++)
    for(int iz = 0; iz<f->get_neta(); iz++)
      for(int ix = 0; ix<f->get_nx(); ix++)
	{      
	  cell *c = f->get_cell(ix, iy, iz );
	  c->save_Q_prev();
	  c->clear_flux();
	}

  // X-direction flux
  for(int iy = 0; iy<f->get_ny(); iy++)
    for(int iz = 0; iz<f->get_neta(); iz++)
      for(int ix = 0; ix<f->get_nx()-1; ix++)
	{
	  hlle_flux(f->get_cell(ix,iy, iz), f->get_cell(ix+1,iy,iz), X_, PREDICT);
	  // cout<<iy<<"\t"<<iz<<"\t"<<ix<<endl;
	}
  
  // Y-direction flux
  for(int iz = 0; iz<f->get_neta(); iz++)
    for(int ix = 0; ix<f->get_nx(); ix++)
      for(int iy = 0; iy<f->get_ny()-1; iy++)
	{
	  hlle_flux(f->get_cell(ix,iy,iz), f->get_cell(ix,iy+1,iz), Y_, PREDICT);
	}
  
if(f->get_neta() > 1) // don,t calculate z-flux in 2+1D hydro
{  
  // Z-direction flux
  for(int ix = 0; ix<f->get_nx(); ix++)
    for(int iy = 0; iy<f->get_ny(); iy++)
      for(int iz = 0; iz<f->get_neta() -1; iz++)
	{
	  hlle_flux(f->get_cell(ix,iy,iz), f->get_cell(ix,iy,iz+1), Z_, PREDICT);
	}
}  
  
  for(int iy = 0; iy<f->get_ny(); iy++)
    for(int iz = 0; iz<f->get_neta(); iz++)
      for(int ix = 0; ix< f->get_nx(); ix++)
        {
	  cell *c = f->get_cell(ix, iy,iz);
	  sourcestep( PREDICT,  ix,  iy, iz,  tau);
	  c->update_Q_to_Qh_by_flux();
	  c->clear_flux();
	}
  
  // X-direction flux
  for(int iy = 0; iy<f->get_ny(); iy++)
    for(int iz = 0; iz<f->get_neta(); iz++)
      for(int ix = 0; ix<f->get_nx()-1; ix++)
	{
	  hlle_flux(f->get_cell(ix,iy, iz), f->get_cell(ix+1,iy,iz), X_, CORRECT);
	}
  
  
  // Y-direction flux
  for(int iz = 0; iz<f->get_neta(); iz++)
    for(int ix = 0; ix<f->get_nx(); ix++)
      for(int iy = 0; iy<f->get_ny()-1; iy++)
	{
	  hlle_flux(f->get_cell(ix,iy,iz), f->get_cell(ix,iy+1,iz), Y_, CORRECT);
	}
  
if(f->get_neta()> 1) // don,t calculate z-flux in 2+1D hydro
{  
  // Z-direction flux
  for(int ix = 0; ix<f->get_nx(); ix++)
    for(int iy = 0; iy<f->get_ny(); iy++)
      for(int iz = 0; iz<f->get_neta()-1; iz++)
	{
	  hlle_flux(f->get_cell(ix,iy,iz), f->get_cell(ix,iy,iz+1), Z_, CORRECT);
	}
  
}  
  for(int iy = 0; iy <f->get_ny(); iy++)
    for(int iz = 0; iz< f->get_neta(); iz++)
      for(int ix = 0; ix< f->get_nx(); ix++)
	{
	  cell *c = f->get_cell(ix, iy,iz); 
	  sourcestep( CORRECT,  ix,  iy, iz, tau);
	  c->update_by_flux();
	  c->clear_flux();	  
	}
  
  tau = tau+dt;                 // tau increased
  f->correct_imaginary_cells(); //boundary condition
  
   
}




void hydro::hlle_flux(cell* left, cell* right, int direction, int mode)
{
  const double dta = mode == 0 ? dt/2. : dt;
  double el,vxl,vyl,vzl,nbl,nql,nsl,pl;
  double er,vxr,vyr,vzr,nbr,nqr,nsr,pr;
  double El,Er;
  double Utl=0.,Uxl=0.,Uyl=0.,Uzl=0.,Ubl=0.,Uql=0.,Usl=0.;
  double Utr=0.,Uxr=0.,Uyr=0.,Uzr=0.,Ubr=0.,Uqr=0.,Usr=0.;
  double Ftl=0.,Fxl=0.,Fyl=0.,Fzl =0,Fbl=0., Fql =0, Fsl =0;
  double Ftr=0.,Fxr=0.,Fyr=0.,Fzr =0,Fbr=0., Fqr =0, Fsr =0;
  double csb,vb,bl=0.,br=0.;
  double flux[7]={0.0};
  double tauFactor;  // fluxes are also multiplied by tau
  double dx =0;
  
  if(mode == PREDICT)
    {
      left->get_right_var(eos, tau, el, pl, nbl,nql,nsl, vxl, vyl, vzl, direction);
      right->get_left_var(eos, tau, er, pr, nbr,nqr,nsr, vxr, vyr, vzr, direction);
      El = (el+pl)*(1.0/(1.0-vxl*vxl-vyl*vyl-vzl*vzl));
      Er = (er+pr)*(1.0/(1.0-vxr*vxr-vyr*vyr-vzr*vzr));
      tauFactor = tau ;
    }
  else
    {
      left->get_right_varH(eos, tau, el, pl, nbl,nql,nsl, vxl, vyl, vzl, direction );
      right->get_left_varH( eos, tau, er, pr, nbr,nqr,nsr, vxr, vyr, vzr, direction);
      El = (el+pl)*(1.0/(1.0-vxl*vxl-vyl*vyl-vzl*vzl));
      Er = (er+pr)*(1.0/(1.0-vxr*vxr-vyr*vyr-vzr*vzr));
      tauFactor = tau +(0.5*dt);
    }
  
  if (el < 0.)
    {
      el = 0.;
      pl = 0.;
    }
  if (er < 0.)
    {
      er = 0.;
      pr = 0.;
    }
  
  if (el < del_eps && er < del_eps ) return;  // *1) no flux calculation if both sides are empty cells
  
  double gammal = 1.0 / sqrt(1 - vxl * vxl - vyl * vyl - vzl*vzl );
  double gammar = 1.0 / sqrt(1 - vxr * vxr - vyr * vyr - vzr*vzr );
  Utl = gammal * gammal * (el + pl) - pl;
  Uxl = gammal * gammal * (el + pl) * vxl;
  Uyl = gammal * gammal * (el + pl) * vyl;
  Uzl = gammal * gammal * (el + pl) * vzl;
  Ubl = gammal * nbl;
  Uql = gammal * nql;
  Usl = gammal * nsl;
  
  
  Utr = gammar * gammar * (er + pr) - pr;
  Uxr = gammar * gammar * (er + pr) * vxr;
  Uyr = gammar * gammar * (er + pr) * vyr;
  Uzr = gammar * gammar * (er + pr) * vzr;
  Ubr = gammar * nbr;
  Uqr = gammar * nqr;
  Usr = gammar * nsr;
  
  
  if(direction == X_)
    {
      Ftl = Utl * vxl + pl * vxl;
      Fxl = Uxl * vxl + pl;
      Fyl = Uyl * vxl;
      Fzl = Uzl * vxl;
      Fbl = Ubl * vxl;
      Fql = Uql * vxl;
      Fsl = Usl * vxl;
      
      Ftr = Utr * vxr + pr * vxr;
      Fxr = Uxr * vxr + pr;
      Fyr = Uyr * vxr;
      Fzr = Uzr * vxr;
      Fbr = Ubr * vxr;
      Fqr = Uqr * vxr;
      Fsr = Usr * vxr;
      
      
      // for the case of constant c_s only
      csb = sqrt(eos->cs2_() +
		 0.5 * sqrt(El * Er) / pow(sqrt(El) + sqrt(Er), 2) *
                 pow(vxl - vxr, 2));

      vb = (sqrt(El) * vxl + sqrt(Er) * vxr) / (sqrt(El) + sqrt(Er));
      bl = min(0., min((vb - csb) / (1 - vb * csb),
		       (vxl - eos->cs_()) / (1 - vxl * eos->cs_())));
      br = max(0., max((vb + csb) / (1 + vb * csb),
		       (vxr + eos->cs_()) / (1 + vxr * eos->cs_())));
      
      dx = f->get_dx();
      
      
      if (el == 0.) bl = -1.; 
      if (er == 0.) br = 1.;
    }
  
  
  
  if(direction == Y_)
    {
      Ftl = Utl * vyl + pl * vyl;
      Fxl = Uxl * vyl ;
      Fyl = Uyl * vyl + pl;
      Fzl = Uzl * vyl;
      Fbl = Ubl * vyl;
      Fql = Uql * vyl;
      Fsl = Usl * vyl;
      
      Ftr = Utr * vyr + pr * vyr;
      Fxr = Uxr * vyr ;
      Fyr = Uyr * vyr + pr;
      Fzr = Uzr * vyr;
      Fbr = Ubr * vyr;
      Fqr = Uqr * vyr;
      Fsr = Usr * vyr;
      
      


      // for the case of constant c_s only
      
     csb = sqrt(eos->cs2_() +
		 0.5 * sqrt(El * Er) / pow(sqrt(El) + sqrt(Er), 2) *
                 pow(vyl - vyr, 2));
     vb = (sqrt(El) * vyl + sqrt(Er) * vyr) / (sqrt(El) + sqrt(Er));
     bl = min(0., min((vb - csb) / (1 - vb * csb),
		      (vyl - eos->cs_()) / (1 - vyl * eos->cs_())));
     br = max(0., max((vb + csb) / (1 + vb * csb),
                   (vyr + eos->cs_()) / (1 + vyr * eos->cs_())));

   
     dx = f->get_dy();
     
     
     if (el == 0.) bl = -1.;
     if (er == 0.) br = 1.;
    }
  
  
  if(direction == Z_)
    {
      double tau1 =tauFactor;
      Ftl = Utl * vzl/tau1 + pl * vzl/tau1;
      Fxl = Uxl * vzl/tau1 ;
      Fyl = Uyl * vzl/tau1;
      Fzl = Uzl * vzl/tau1 + pl/tau1;
      Fbl = Ubl * vzl/tau1;
      Fql = Uql * vzl/tau1;
      Fsl = Usl * vzl/tau1;
      
      Ftr = Utr * vzr/tau1 + pr * vzr/tau1;
      Fxr = Uxr * vzr/tau1 ;
      Fyr = Uyr * vzr/tau1 ;
      Fzr = Uzr * vzr/tau1 + pr/tau1 ;
      Fbr = Ubr * vzr/tau1;
      Fqr = Uqr * vzr/tau1;
      Fsr = Usr * vzr/tau1;
      
      
      
    // for the case of constant c_s only
     
     csb = sqrt(eos->cs2_() +
		 0.5 * sqrt(El * Er) / pow(sqrt(El) + sqrt(Er), 2) *
                 pow(vzl - vzr, 2));

     vb = (sqrt(El) * vzl + sqrt(Er) * vzr) / (sqrt(El) + sqrt(Er));
     bl = (1.0/tau)*min(0., min((vb - csb) / (1 - vb * csb),
				(vzl - eos->cs_()) / (1 - vzl * eos->cs_())));
     br = (1.0/tau)*max(0., max((vb + csb) / (1 + vb * csb),
				 (vzr + eos->cs_()) / (1 + vzr * eos->cs_())));

    


     dx =f->get_deta();
     
     
    if (el == 0.) bl = -1./tau; 
    if (er == 0.) br = 1./tau;
    }
  

  if(bl == 0. && br == 0.) return;
  
  flux[T_] = tauFactor * dta / dx *
    (-bl * br * (Utl - Utr) + br * Ftl - bl * Ftr) / (-bl + br);
  flux[X_] = tauFactor * dta / dx *
    (-bl * br * (Uxl - Uxr) + br * Fxl - bl * Fxr) / (-bl + br);
  flux[Y_] = tauFactor * dta / dx *
    (-bl * br * (Uyl - Uyr) + br * Fyl - bl * Fyr) / (-bl + br);
  flux[Z_] = tauFactor * dta / dx *
    (-bl * br * (Uzl - Uzr) + br * Fzl - bl * Fzr) / (-bl + br);
  flux[NB_] = tauFactor * dta / dx *
    (-bl * br * (Ubl - Ubr) + br * Fbl - bl * Fbr) / (-bl + br);
  flux[NQ_] = tauFactor * dta / dx *
    (-bl * br * (Uql - Uqr) + br * Fql - bl * Fqr) / (-bl + br);
  flux[NS_] = tauFactor * dta / dx *
    (-bl * br * (Usl - Usr) + br * Fsl - bl * Fsr) / (-bl + br);
  
  
  left->add_flux(-flux[T_], -flux[X_], -flux[Y_], -flux[Z_], -flux[NB_],
		-flux[NQ_], -flux[NS_]);
  right->add_flux(flux[T_], flux[X_], flux[Y_], flux[Z_], flux[NB_], flux[NQ_],
		 flux[NS_]);

}




void hydro::sourcestep(int mode, int ix, int iy, int iz, double _tau)
{ 
  double S[7];
  double e,p,vx,vy,vz,nb,nq,ns;
  if( mode == PREDICT )
    {
      double _dt  = 0.5*dt; // (1/2)*dtau -> for predictor step 
      f->get_cell(ix,iy,iz)->get_Q(S);
      for(int i =0; i<7; i++){S[i] = S[i]/_tau; }
      CN->CALC_2_LRF(eos,  S, e, p, nb, nq, ns, vx, vy, vz);
      if (e < del_eps ) return;
      f->get_cell(ix,iy,iz)->add_flux( (-S[T_] * vz * vz - p * (1. + vz * vz))*_dt, 0.0, 0.0, -S[Z_]*_dt, 0.0,0.0, 0.0);  
    }
  else
    {
      _tau = _tau+0.5*dt;
      double _dt  = dt;
      f->get_cell(ix,iy,iz)->get_Qh(S);
      for(int i =0; i<7; i++){S[i] = S[i]/_tau; }
      CN->CALC_2_LRF(eos,  S, e, p, nb, nq, ns, vx, vy, vz);
      if (e < del_eps) return;
      f->get_cell(ix,iy,iz)->add_flux(  (-S[T_] * vz * vz - p * (1. + vz * vz))*_dt, 0.0, 0.0, (-S[Z_])*_dt, 0.0,0.0, 0.0);     
    }
  
}








