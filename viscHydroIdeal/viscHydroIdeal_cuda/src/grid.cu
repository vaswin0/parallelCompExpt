#include "hip/hip_runtime.h"
#include "grid.h"
#include "global.h"

grid::grid(idb* _IDB, cnvrt* _CN, EoS* _eos)
{
  IDB = _IDB ;
  CN = _CN;
  eos = _eos;
}

grid::~grid()
{
// delete[] Cell; 
hipFree(Cell);
}

void grid::make_grid()
{
  
  if(IDB->ic_mode == 2)
    {
      double dummy2;
      string dummy;
      double temp_nx, temp_ny ; // x and y from file.
      double xf, yf;
      int neta_;
      double deta_,dx_,dy_;
      
      
      std::fstream ic_file;
      ic_file.open(IDB->init_file_name.c_str(),ios::in);
      if (!ic_file) 
	{
	  cout<<"couldn't find ic file."<<endl;
	  exit(1);
	}
      
      ic_file.getline(buff,200);
      
      iss = new istringstream(buff);
      
      *iss >> dummy >> dummy >> dummy2 
	   >> dummy >> neta_ >> dummy >> temp_nx >> dummy >> temp_ny 
	   >> dummy >> deta_ >> dummy >> dx_ >> dummy >> dy_ ;
      
      IDB->nx = temp_nx ;
      IDB->ny = temp_ny ;
      
      delete iss;
      
      ic_file.getline(buff,200);
      iss = new istringstream(buff);
      *iss >> dummy2 >> xf >> yf 
	   >> dummy2 >> dummy2 >> dummy2 >> dummy2
	   >> dummy2  >> dummy2  >> dummy2  >> dummy2;
      
      delete iss;
      IDB->xmin = xf ;
      IDB->xmax = fabs(xf) ;
      IDB->ymin = yf ;
      IDB->ymax = fabs(yf) ;
      
      IDB->dx =  ( IDB->xmax- IDB->xmin )
	         /  ( IDB->nx - 1 ) ;
      IDB->dy =  ( IDB->ymax- IDB->ymin )
	         / ( IDB->ny - 1 ) ;
      
      cout<<"[Info] grid resized."<<endl;
      cout<<"[Info] xmin : "<<IDB->xmin<<"  xmax : "<<IDB->xmax<<endl;
      cout<<"[Info] ymin : "<<IDB->ymin<<"  ymax : "<<IDB->ymax<<endl;
      cout<<"[Info] dx : "<<IDB->dx<<endl;
      cout<<"[Info] dy : "<<IDB->dy<<endl;
      
    }


  if (IDB->dtau > IDB->dx / 2. ||
      IDB->dtau > IDB->dy / 2. )
     {
      cout << "[Error]  too big delta_tau : " << IDB->dtau <<
              "  " << IDB->dx << "  " << IDB->dtau << endl;
      exit(1);
     }

 if(IDB->neta > 1 && 
    IDB->dtau > IDB->tau0*IDB->deta/2. )
   {
     cout << "[Error]  too big delta_tau : " << IDB->dtau <<
          "  tau*deta/2. = "<< IDB->tau0*IDB->deta/2. << endl; 
     exit(1);
   }
  
  
  cout<<"[Info] nx = "<<IDB->nx<<endl;
  cout<<"[Info] ny = "<<IDB->ny<<endl;
  cout<<"[info] neta = "<<IDB->neta<<"\n"<<endl;



  nx = IDB->nx ;
  ny = IDB->ny ;
  neta = IDB->neta ;

  dx = IDB->dx ;
  dy = IDB->dy ;
  deta = IDB->deta ;

  xmin = IDB->xmin;
  xmax = IDB->xmax;
  ymin = IDB->ymin;
  ymax = IDB->ymax;
  etamin = IDB->etamin;
  etamax = IDB->etamax;
  
  dtau = IDB->dtau ;
  
  
  
  // now grid properties are fixed and it's ready to be made
  Cell = new cell[nx * ny * neta];



/*  
  for(int i = 0; i < IDB->nx; i++)
    {
      vector < vector < cell > > w;
      tube.push_back( w );
      for(int j = 0; j < IDB->ny; j++)
	{
	  vector <cell> v;
	  tube[i].push_back( v );
	  for(int k = 0; k < IDB->neta; k++)
	    {
	      //cout<<"i = "<<i << " j = " << j << " k = " << k << endl;
	      tube[i][j].push_back(*new_cell(i,j,k));
	      //c1->set_pos(i,j,k);
	      //cout<<new_cell(i,j,k)<<endl;
	    }
	}
    }
*/ 
  
  
  for (int iz = 0; iz < IDB->neta; iz++)
    for (int iy = 0; iy < IDB->ny; iy++)
      for (int ix = 0; ix < IDB->nx; ix++)
	{
          //cout<<"iy = "<<iy << " ix = " << ix <<endl;
          get_cell(ix, iy, iz)->set_pos(ix,iy,iz);
          get_cell(ix, iy, iz)->set_cnvrt(CN);
	  get_cell(ix, iy, iz)->set_prev_cell(X_, get_cell(ix - 1, iy, iz));
	  get_cell(ix, iy, iz)->set_next_cell(X_, get_cell(ix + 1, iy, iz));
	  get_cell(ix, iy, iz)->set_prev_cell(Y_, get_cell(ix, iy - 1, iz));
	  get_cell(ix, iy, iz)->set_next_cell(Y_, get_cell(ix, iy + 1, iz));
	  get_cell(ix, iy, iz)->set_prev_cell(Z_, get_cell(ix, iy, iz - 1));
	  get_cell(ix, iy, iz)->set_next_cell(Z_, get_cell(ix, iy, iz + 1));    
	}
  
}


cell* grid::new_cell(int ix,int iy,int iz)
{
  cell* c1 = new cell();
  c1->set_pos(ix,iy,iz);
  return c1;
}





void grid::correct_imaginary_cells()
{

  double Q[7]={0.0};
  // Z
 for (int ix = 0; ix < IDB->nx; ix++)
  for (int iy = 0; iy < IDB->ny; iy++) {
   // left boundary
   get_cell(ix, iy, 2)->get_Q(Q);
   get_cell(ix, iy, 1)->set_Q(Q);
   get_cell(ix, iy, 0)->set_Q(Q);
   // right boundary
   get_cell(ix, iy, IDB->neta - 3)->get_Q(Q);
   get_cell(ix, iy, IDB->neta - 2)->set_Q(Q);
   get_cell(ix, iy, IDB->neta - 1)->set_Q(Q);
  }
 // Y
 for (int ix = 0; ix < IDB->nx; ix++)
  for (int iz = 0; iz < IDB->neta; iz++) {
   // left boundary
   get_cell(ix, 2, iz)->get_Q(Q);
   get_cell(ix, 1, iz)->set_Q(Q);
   get_cell(ix, 0, iz)->set_Q(Q);
   // right boundary
   get_cell(ix, IDB->ny - 3, iz)->get_Q(Q);
   get_cell(ix, IDB->ny - 2, iz)->set_Q(Q);
   get_cell(ix, IDB->ny - 1, iz)->set_Q(Q);
  }
 // X
 for (int iy = 0; iy < IDB->ny; iy++)
  for (int iz = 0; iz < IDB->neta; iz++) {
   // left boundary
   get_cell(2, iy, iz)->get_Q(Q);
   get_cell(1, iy, iz)->set_Q(Q);
   get_cell(0, iy, iz)->set_Q(Q);
   // right boundary
   get_cell(IDB->nx - 3, iy, iz)->get_Q(Q);
   get_cell(IDB->nx - 2, iy, iz)->set_Q(Q);
   get_cell(IDB->nx - 1, iy, iz)->set_Q(Q);
  }
 
}



void grid::correct_imaginary_cells_full() {
 double Q[7], _pi[4][4], _Pi;
 // Z
 for (int ix = 0; ix < IDB->nx; ix++)
  for (int iy = 0; iy < IDB->ny; iy++) {
   // left boundary
   get_cell(ix, iy, 2)->get_Q(Q);
   get_cell(ix, iy, 1)->set_Q(Q);
   get_cell(ix, iy, 0)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++) _pi[i][j] = get_cell(ix, iy, 2)->get_pi(i, j);
   _Pi = get_cell(ix, iy, 2)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(ix, iy, 0)->set_pi(i, j, _pi[i][j]);
     get_cell(ix, iy, 1)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(ix, iy, 0)->set_Pi(_Pi);
   get_cell(ix, iy, 1)->set_Pi(_Pi);
   // right boundary
   get_cell(ix, iy, IDB->neta - 3)->get_Q(Q);
   get_cell(ix, iy, IDB->neta - 2)->set_Q(Q);
   get_cell(ix, iy, IDB->neta - 1)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
     _pi[i][j] = get_cell(ix, iy, IDB->neta - 3)->get_pi(i, j);
   _Pi = get_cell(ix, iy, IDB->neta - 3)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(ix, iy, IDB->neta - 2)->set_pi(i, j, _pi[i][j]);
     get_cell(ix, iy, IDB->neta - 1)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(ix, iy, IDB->neta - 2)->set_Pi(_Pi);
   get_cell(ix, iy, IDB->neta - 1)->set_Pi(_Pi);
  }
 // Y
 for (int ix = 0; ix < IDB->nx; ix++)
  for (int iz = 0; iz < IDB->neta; iz++) {
   // left boundary
   get_cell(ix, 2, iz)->get_Q(Q);
   get_cell(ix, 1, iz)->set_Q(Q);
   get_cell(ix, 0, iz)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++) _pi[i][j] = get_cell(ix, 2, iz)->get_pi(i, j);
   _Pi = get_cell(ix, 2, iz)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(ix, 0, iz)->set_pi(i, j, _pi[i][j]);
     get_cell(ix, 1, iz)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(ix, 0, iz)->set_Pi(_Pi);
   get_cell(ix, 1, iz)->set_Pi(_Pi);
   // right boundary
   get_cell(ix, IDB->ny - 3, iz)->get_Q(Q);
   get_cell(ix, IDB->ny - 2, iz)->set_Q(Q);
   get_cell(ix, IDB->ny - 1, iz)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
     _pi[i][j] = get_cell(ix, IDB->ny - 3, iz)->get_pi(i, j);
   _Pi = get_cell(ix, IDB->ny - 3, iz)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(ix, IDB->ny - 2, iz)->set_pi(i, j, _pi[i][j]);
     get_cell(ix, IDB->ny - 1, iz)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(ix, IDB->ny - 2, iz)->set_Pi(_Pi);
   get_cell(ix, IDB->ny - 1, iz)->set_Pi(_Pi);
  }
 // X
 for (int iy = 0; iy < IDB->ny; iy++)
  for (int iz = 0; iz < IDB->neta; iz++) {
   // left boundary
   get_cell(2, iy, iz)->get_Q(Q);
   get_cell(1, iy, iz)->set_Q(Q);
   get_cell(0, iy, iz)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++) _pi[i][j] = get_cell(2, iy, iz)->get_pi(i, j);
   _Pi = get_cell(2, iy, iz)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(0, iy, iz)->set_pi(i, j, _pi[i][j]);
     get_cell(1, iy, iz)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(0, iy, iz)->set_Pi(_Pi);
   get_cell(1, iy, iz)->set_Pi(_Pi);
   // right boundary
   get_cell(IDB->nx - 3, iy, iz)->get_Q(Q);
   get_cell(IDB->nx - 2, iy, iz)->set_Q(Q);
   get_cell(IDB->nx - 1, iy, iz)->set_Q(Q);
   for (int i = 0; i < 4; i++)
    for (int j = 0; j < 4; j++)
     _pi[i][j] = get_cell(IDB->nx - 3, iy, iz)->get_pi(i, j);
   _Pi = get_cell(IDB->nx - 3, iy, iz)->get_Pi();

   for (int i = 0; i < 4; i++)
    for (int j = 0; j <= i; j++) {
     get_cell(IDB->nx - 2, iy, iz)->set_pi(i, j, _pi[i][j]);
     get_cell(IDB->nx - 1, iy, iz)->set_pi(i, j, _pi[i][j]);
    }
   get_cell(IDB->nx - 2, iy, iz)->set_Pi(_Pi);
   get_cell(IDB->nx - 1, iy, iz)->set_Pi(_Pi);
  }
}






