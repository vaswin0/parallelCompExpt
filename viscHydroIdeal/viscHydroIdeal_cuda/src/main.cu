#include <iostream>
#include <iomanip>
#include <string>
#include <sstream>
#include <chrono>
#include <cmath>
#include "idb.h"
#include "read_id.h"
#include "master.h"

using std::cout;
using std::endl;
using namespace std::chrono;


int main(int argc, char **argv)
{
  
  // reading the input data from the file
  string input_file_name;char* event_no_s ;

  if(argc == 3){
      event_no_s = *(argv+1); input_file_name = *(argv+2);}
  else
     {
      cout<<"[Info] Please give 2 arguments\n"
            "1st argument - event no\n    "
            "2nd argument - input filename"<<endl;
      exit(1);
     }

  read_id* reader = new read_id(); 
  idb *IDB = new idb;
  reader->read_id_from_file(IDB, input_file_name); // read input data base and store it.
  int event_no = atof(event_no_s) ;

  cout<<"event no : "<<event_no<<endl;


  master head =  master(IDB);
  head.initialize();
  master *pHd;
  hipMallocManaged(&pHd, sizeof(evolve));
  *pHd = head



  //head->initialize(); // initialize
  head->run_hydro(); // run hydro 
 

 

 delete head;
 return 0;
}



