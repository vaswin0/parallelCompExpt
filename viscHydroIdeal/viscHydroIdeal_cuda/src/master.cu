#include "hip/hip_runtime.h"
#include "master.h"

master::master(idb* _IDB)
{
  // Creating the "hydro_output" directory if it doesn't exist
  if (mkdir("outputs", 0777) == -1)
    cerr << "[Info] " << strerror(errno) << endl;
  else
    cout << "[Info] hydro_output directory is created" << endl;

  IDB = _IDB;
//  eos = new EoS0();
  hipMallocManaged(&eos, sizeof(Eos0));
  //CN = new cnvrt();
  hipMallocManaged(&CN, sizeof(cnvrt)

  
}

master::~master()
{
  delete IDB;
 // delete eos;
 hipFree(eos);
  delete h;
  //delete g;
 // delete CN;
 hipFree(g);
 hipFree(CN);
}

void master::initialize()
{

  // make the grid
  g = new grid(IDB,CN,eos);
  g->make_grid();

  ic = new init(IDB);
  ic->set_init(g,eos);
    
}



void master::run_hydro(){

  double current_tau ; 
  
  h = new hydro(eos, g , IDB , IDB->tau0 , IDB->dtau, CN);
  int nstep = (IDB->tauMax - IDB->tau0)/IDB->dtau + 1;
  std::cout << "Hydro evolution ..." << std::endl ;
  std::cout << "evolution till max. tau : " << IDB->tauMax << std::endl ;   
  for(int istep=0; istep<nstep ; istep++){
    current_tau =  IDB->tau0 + istep * IDB->dtau ;
  
    if(istep%10==0){ 
    std::cout << "tau : " << current_tau << std::endl ;
    } 

    if(istep%100 == 0 ){
      write_grid_info(current_tau) ; 
    }

    // entire hydro evolution here
    h->evolve();      
  }// step loop
}




void master::write_grid_info(double t){

  std::cout << "storing fluid information at tau :"
       << t << " (fm)" << std::endl ; 

  std::ofstream mFile;
  std::stringstream output_filename;

  output_filename.str("");
  output_filename << "outputs/fluid_info_at_tau_" << t << ".txt";

  mFile.open(output_filename.str().c_str(), std::ios::out );
  double epsilon, pressure, vx, vy, vz, nb, nq, ns ; 
  for(int ix=0; ix<IDB->nx; ix++){
    for(int iy=0; iy<IDB->ny; iy++){
       double xx = g->get_x(ix);;
       double yy = g->get_y(iy);;
       double rr = sqrt(xx*xx+yy*yy);
       g->get_cell(ix,iy,0)->get_physical_var(eos, t, epsilon, pressure, nb, nq, ns, vx, vy, vz); 
       mFile << xx << "  " << yy << "  " << rr << "  " << epsilon << "  " << nb << "  " << vx << "  " << vy << std::endl ; 
    }
  }

  mFile.close() ; 
}


















