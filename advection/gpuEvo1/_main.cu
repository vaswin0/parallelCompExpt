#include "hip/hip_runtime.h"
#include <iostream>
//#include "eos.h"
#include "cell.h"
#include "grid.h"
#include "init.h"
#include "evolve.h"
using namespace std;
const int N = 50;
//__device__ grid Gr = grid(-12,12,50,0,0,1,0,0,1);


__global__ void funct(cell* c, grid* g){

g->Cell = c;

printf("%f", c[1].get_rho());
}


int main(){
	
	int nx = N;
	int ny = 1;
	int nz = 1;

	cell *pCell;
	
	size_t cellSize = nx*ny*nz*sizeof(cell);

	hipMalloc(&pCell, cellSize); // cout<<"hipMalloc(&pCell, cellSize) worked"<<endl;

	grid Gr = grid(-12,12,N,0,0,1,0,0,1);
//	cout << Gr.Cell<<"from host" <<endl;

	grid* pGr_d;

	
	init *in = new init(&Gr);// cout<<"made init obj"<<endl;
	in->set_init(); //cout<< "initialized" <<endl;
//	cout<<Gr.Cell[1].get_rho()<<endl;

	hipMemcpy(pCell, &Gr.Cell, cellSize, hipMemcpyHostToDevice);
	
	size_t gridSize = sizeof(grid);
	hipMalloc(&pGr_d, gridSize);
	hipMemcpy(pGr_d, &Gr, gridSize, hipMemcpyHostToDevice);

	evolve Ev = evolve(pGr_d, 1,0.01);
	evolve* pEv_d;
	size_t evosz = sizeof(Ev);
	hipMalloc(&pEv_d, evosz);
	hipMemcpy(pEv_d, &Ev, evosz, hipMemcpyHostToDevice);

	funct<<<1,1>>>(pCell, pGr_d);
	hipDeviceSynchronize();	
	   
	return 0;



}
