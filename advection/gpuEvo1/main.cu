#include <iostream>
//#include "eos.h"
#include "cell.h"
#include "grid.h"
#include "init.h"
#include "evolve.h"
using namespace std;
const int N = 50;
//__device__ grid Gr = grid(-12,12,50,0,0,1,0,0,1);

int main(){
	
	int nx = N;
	int ny = 1;
	int nz = 1;

	cell *pCell;
	


	
	size_t cellSize = nx*ny*nz*sizeof(cell);

	hipMalloc(&pCell, cellSize);  cout<<"hipMalloc(&pCell, cellSize) worked"<<endl;

	grid Gr = grid(-12,12,N,0,0,1,0,0,1);
	cout << Gr.Cell <<endl;
	Gr.Cell = pCell; cout<<Gr.Cell<<endl;
	
	grid *pGr_d;

	
	std::cout<<4;

	init *in = new init(&Gr); cout<<"made init obj"<<endl;
	in->set_init(); cout<< "initialized" <<endl;
//	std::cout<<Gr<<std::endl;
 	std::cout<<5;

//	evolve Ev = evolve(pGr_d, 1.0,0.01);


	size_t gridSize = sizeof(grid);
	hipMalloc(&pGr_d, gridSize);
	hipMemcpy(pGr_d, &Gr,gridSize, hipMemcpyHostToDevice);
	
	evolve Ev = evolve(pGr_d, 1.0,0.01);
	evolve* pEv_d;
	size_t evoSz = sizeof(Ev);
	hipMalloc(&pEv_d, evoSz);
	hipMemcpy(pEv_d, &Ev, evoSz, hipMemcpyHostToDevice);


//	evolvee<<<1,1>>>(pEv_d);
	//std::cout<<122;
//	hipDeviceSynchronize();
	
		   
	return 0;



}
