#include "hip/hip_runtime.h"
#include <iostream>
//#include "eos.h"
#include "cell.h"
#include "grid.h"
#include "init.h"
#include "evolve.h"
using namespace std;
const int N = 50;
//__device__ grid Gr = grid(-12,12,50,0,0,1,0,0,1);


__global__ void initialize(evolve* Ev){

	grid* gr = new grid (-12,12,50,0,0,1,0,0,1);
	init* in = new init(gr); in->set_init();
	Ev = new evolve(gr, 1.0,0.01);




}

__global__ void evolvee(evolve* Ev){


	int i = blockDim.x*blockIdx.x + threadIdx.x;
	if (i < N) {Ev->calc_flux(i,0,0);printf("%d", i);};
}

int main(){
	
	
	evolve* pEv;
	hipMalloc(&pEv, sizeof(evolve));


	initialize<<<1,1>>>(pEv);

	hipDeviceSynchronize();	

	hipMemcpy(var_d, var, sizeof(int), hipMemcpyDeviceToDevice);

	
	evolvee<<<5,10>>>(pEv);
	hipDeviceSynchronize();
	   
	return 0;



}
