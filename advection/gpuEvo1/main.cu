#include "hip/hip_runtime.h"
#include <iostream>
//#include "eos.h"
#include "cell.h"
#include "grid.h"
#include "init.h"
#include "evolve.h"

const int N = 50;

__device__ grid Gr;
__device__ init in;
__device__ evolve Ev;



__global__ void evolvee(){
   int i= blockIdx.x*blockDim.x + threadIdx.x;
   if(i<N){
		        
        Ev.calc_flux(i, 0,0);
		__syncthreads();
		Ev.getCellUpdateRho(i, 0,0);
		__syncthreads();
		Ev.getCellClearFlux(i,0,0);

            }
        }			




int main(){

	  double xmin = -12 ; 
	  double xmax =  12 ;
	  int    nx   = N;  
	  double ymin = 0 ; 
	  double ymax = 0 ;
	  int    ny   = 1 ;  
	  double zmin = 0 ; 
	  double zmax = 0 ;
	  int    nz   = 1 ;  

	  grid  Gr = grid(xmin, xmax, nx, ymin, ymax, ny, zmin, zmax, nz );
	  
	  grid* pGr;

	 hipMallocManaged(&pGr, sizeof(grid));
	 *pGr = Gr;


	init *in = new init(pGr);
	in->set_init();

	std::cout<<Gr<<std::endl;



	evolve Ev = evolve(pGr, 1.0,0.01);
	evolve* pEv;
	hipMallocManaged(&pEv, sizeof(evolve));
	*pEv = Ev;

	for(int i = 0; i < 100; ++i){
	evolvee<<<4,15>>>(pEv);
	hipDeviceSynchronize();
	}
		    //pR->displayArray();

	std::cout<<Gr;
	return 0;



}
