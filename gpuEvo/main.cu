#include "hip/hip_runtime.h"
#include <iostream>
//#include "eos.h"
#include "cell.h"
#include "grid.h"
#include "init.h"
#include "evolve.h"

const int N = 1000000;


__global__ void evolvee(evolve *Ev){
   int i= blockIdx.x*blockDim.x + threadIdx.x;
   if(i<N){
		        
        Ev->calc_flux(i, 0,0);
		__syncthreads();
		Ev->getCellUpdateRho(i, 0,0);
		__syncthreads();
		Ev->getCellClearFlux(i,0,0);

            }
        }			




int main(){

  double xmin = -12 ; 
  double xmax =  12 ;
  int    nx   = N;  
  double ymin = 0 ; 
  double ymax = 0 ;
  int    ny   = 1 ;  
  double zmin = 0 ; 
  double zmax = 0 ;
  int    nz   = 1 ;  

grid  Gr = grid(xmin, xmax, nx, ymin, ymax, ny, zmin, zmax, nz );
  
 grid* pGr;

 hipMallocManaged(&pGr, sizeof(grid));
 *pGr = Gr;


init *in = new init(pGr);
in->set_init();

std::cout<<Gr<<std::endl;



evolve Ev = evolve(pGr, 1.0,0.01);
evolve* pEv;
hipMallocManaged(&pEv, sizeof(evolve));
*pEv = Ev;


evolvee<<<4,26>>>(pEv);
hipDeviceSynchronize();
            //pR->displayArray();

std::cout<<Gr;
return 0;



}
